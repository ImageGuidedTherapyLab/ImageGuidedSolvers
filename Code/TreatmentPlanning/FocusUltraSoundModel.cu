#include "hip/hip_runtime.h"
static char help[] = "Solves -Laplacian u - exp(u) = 0,  0 < x < 1 using GPU\n\n";
/*
   Same as ex47.c except it also uses the GPU to evaluate the function
*/

#include <petscdmda.h>
#include <petscsnes.h>
#include <petsccusp.h>
#include "cusp/detail/device/utils.h"
//#include <thrust/sort.h>

extern PetscErrorCode ComputeFunction(SNES,Vec,Vec,void*);
PetscBool  useCUSP = PETSC_FALSE;
PetscBool  jacobianComputed = PETSC_FALSE;
PetscLogEvent LogFunction = 0;
__device__ PetscInt *cudaTest;

struct LinearHexMesh
{
  LinearHexMesh(PetscInt numelements) : 
              m_NumElements (numelements)
               {
                  m_NodesPerElement = 8 ; 
                  element_residuals->resize(numelements*m_NodesPerElement); 
               }
  // number of elements
  PetscInt m_NumElements;
  // number of nodes per element
  PetscInt m_NodesPerElement  ; 
  // node coordinates
  CUSPARRAY *m_NodeXCoord, *m_NodeYCoord, *m_NodeZCoord;
  // solution and residual
  CUSPARRAY *uarray,*farray;
  // temporary vector to hold element wise residual
  // 8 residual entries per element (one for each node)
  CUSPARRAY *element_residuals;
  // connectivity information is stored per node for structure of array access
  CUSPINTARRAYGPU  *m_Connectivity0,
                   *m_Connectivity1,
                   *m_Connectivity2,
                   *m_Connectivity3,
                   *m_Connectivity4,
                   *m_Connectivity5,
                   *m_Connectivity6,
                   *m_Connectivity7;
  CUSPINTARRAYGPU  *m_GlobalLocalMap0,
                   *m_GlobalLocalMap1,
                   *m_GlobalLocalMap2,
                   *m_GlobalLocalMap3,
                   *m_GlobalLocalMap4,
                   *m_GlobalLocalMap5,
                   *m_GlobalLocalMap6,
                   *m_GlobalLocalMap7;
  CUSPINTARRAYGPU  *m_LocalElementMap;
  typedef CUSPARRAY::iterator PetscScalarIter;
  typedef CUSPINTARRAYGPU::iterator    PetscIntIter;
  typedef thrust::permutation_iterator<PetscScalarIter,PetscIntIter> PetscMapIter;
  typedef thrust::zip_iterator< thrust::tuple< 
     PetscMapIter, PetscMapIter, PetscMapIter, PetscMapIter,
     PetscMapIter, PetscMapIter, PetscMapIter, PetscMapIter 
                       > > hex_iterator ; 
  // iterators for looping of nodes within elements
  typedef thrust::zip_iterator<
            thrust::tuple< hex_iterator, hex_iterator, hex_iterator > > hex_node_iterator;
  hex_node_iterator ElementBegin() 
  { 
    return thrust::make_zip_iterator(thrust::make_tuple( 
             thrust::make_zip_iterator(thrust::make_tuple(   // x - coordinates 
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity0->begin()),
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity1->begin()),
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity2->begin()),
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity3->begin()),
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity4->begin()),
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity5->begin()),
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity6->begin()),
               thrust::make_permutation_iterator(m_NodeXCoord->begin(),m_Connectivity7->begin()) 
                                                         )),
             thrust::make_zip_iterator(thrust::make_tuple(   // y - coordinates 
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity0->begin()),
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity1->begin()),
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity2->begin()),
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity3->begin()),
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity4->begin()),
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity5->begin()),
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity6->begin()),
               thrust::make_permutation_iterator(m_NodeYCoord->begin(),m_Connectivity7->begin()) 
                                                         )),
             thrust::make_zip_iterator(thrust::make_tuple(   // z - coordinates 
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity0->begin()),
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity1->begin()),
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity2->begin()),
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity3->begin()),
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity4->begin()),
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity5->begin()),
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity6->begin()),
               thrust::make_permutation_iterator(m_NodeZCoord->begin(),m_Connectivity7->begin()) 
                                                         )) 
                                                       ));
  } 
  hex_node_iterator ElementEnd() 
  { 
    return thrust::make_zip_iterator(thrust::make_tuple( 
             thrust::make_zip_iterator(thrust::make_tuple(   // x - coordinates 
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity0->end()),
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity1->end()),
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity2->end()),
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity3->end()),
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity4->end()),
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity5->end()),
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity6->end()),
               thrust::make_permutation_iterator(m_NodeXCoord->end(),m_Connectivity7->end()) 
                                                         )),
             thrust::make_zip_iterator(thrust::make_tuple(   // y - coordinates 
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity0->end()),
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity1->end()),
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity2->end()),
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity3->end()),
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity4->end()),
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity5->end()),
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity6->end()),
               thrust::make_permutation_iterator(m_NodeYCoord->end(),m_Connectivity7->end()) 
                                                         )),
             thrust::make_zip_iterator(thrust::make_tuple(   // z - coordinates 
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity0->end()),
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity1->end()),
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity2->end()),
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity3->end()),
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity4->end()),
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity5->end()),
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity6->end()),
               thrust::make_permutation_iterator(m_NodeZCoord->end(),m_Connectivity7->end()) 
                                                         )) 
                                                       ));
  } 
  // iterators for looping over element wise residual for each element
  typedef thrust::zip_iterator< thrust::tuple< 
     PetscScalarIter, PetscScalarIter, PetscScalarIter, PetscScalarIter,
     PetscScalarIter, PetscScalarIter, PetscScalarIter, PetscScalarIter 
                       > > residual_iterator ; 
  residual_iterator ResidualBegin() 
  { 
    return thrust::make_zip_iterator(thrust::make_tuple( 
                                   element_residuals->begin()+0,
                                   element_residuals->begin()+1,
                                   element_residuals->begin()+2,
                                   element_residuals->begin()+3,
                                   element_residuals->begin()+4,
                                   element_residuals->begin()+5,
                                   element_residuals->begin()+6,
                                   element_residuals->begin()+7
                                                       ));
  } 
  residual_iterator ResidualEnd() 
  { 
    return thrust::make_zip_iterator(thrust::make_tuple( 
                                   element_residuals->end()+0,
                                   element_residuals->end()+1,
                                   element_residuals->end()+2,
                                   element_residuals->end()+3,
                                   element_residuals->end()+4,
                                   element_residuals->end()+5,
                                   element_residuals->end()+6,
                                   element_residuals->end()+7
                                                       ));
  } 

  // iterators for looping over element solution vector for each element
  hex_iterator SolutionBegin() 
  {
    return thrust::make_zip_iterator(thrust::make_tuple( 
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap0->begin()),
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap1->begin()),
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap2->begin()),
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap3->begin()),
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap4->begin()),
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap5->begin()),
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap6->begin()),
          thrust::make_permutation_iterator(uarray->begin(),m_GlobalLocalMap7->begin())
                                                       ));
  }
  hex_iterator SolutionEnd() 
  {
    return thrust::make_zip_iterator(thrust::make_tuple( 
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap0->end()),
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap1->end()),
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap2->end()),
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap3->end()),
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap4->end()),
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap5->end()),
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap6->end()),
          thrust::make_permutation_iterator(uarray->end(),m_GlobalLocalMap7->end())
                                                       ));
  }

  typedef thrust::tuple< 
     PetscScalar,PetscScalar,PetscScalar,PetscScalar,
     PetscScalar,PetscScalar,PetscScalar,PetscScalar
  > hex_tuple; 
  // helper to get array/vector from hex tuple
  __host__ __device__
  void get_hex_vector(hex_tuple const &tuple, PetscScalar elemvector[8]) 
  { 
        // decode the tuple
        elemvector[0]=    thrust::get<0>(tuple) ;
        elemvector[1]=    thrust::get<1>(tuple) ;
        elemvector[2]=    thrust::get<2>(tuple) ;
        elemvector[3]=    thrust::get<3>(tuple) ;
        elemvector[4]=    thrust::get<4>(tuple) ;
        elemvector[5]=    thrust::get<5>(tuple) ;
        elemvector[6]=    thrust::get<6>(tuple) ;
        elemvector[7]=    thrust::get<7>(tuple) ;
        return;
  } 
};
// https://groups.google.com/forum/?fromgroups=#!topic/thrust-users/mqYDi2X7xmA
//
// An object's data members exist wherever the compiler decides to place
// them, given some constraints.  For functors used with Thrust, data
// members get copied around to different memory spaces.  A functor (and
// its data) begin on the host, probably implemented by the compiler in
// CPU registers.  A Thrust algorithm will receive a copy of the user's
// functor and eventually package it up in something passed as a
// __global__ function argument.  Depending on various particulars of the
// compiler, GPU, and size, __global__ function arguments may be
// implemented in either __shared__ memory, __constant__ memory, or
// global device memory.  When a __global__ function executes, its
// parameters (including any copies of user functors) typically get
// copied into GPU registers.  Does that make sense?
struct WFSModel : public LinearHexMesh
{
  PetscInt       m_rank,m_deviceNum; //device info
  PetscScalar    m_x0,m_y0,m_z0;
  PetscScalar    m_density           ;
  PetscScalar    m_specificheat      ;
  PetscScalar    m_deltat            ;
  PetscScalar    m_bloodspecificheat ;
  PetscScalar    m_bodytemp          ;
  CUSPARRAY      *m_conduction       ;
  CUSPARRAY      *m_perfusion        ;
  CUSPARRAY      *m_absorption       ;
  CUSPARRAY      *m_scattering       ;
  
  WFSModel(PetscInt rank, PetscInt deviceNum,PetscInt numelements ) : 
             LinearHexMesh(numelements) ,
              m_rank(rank),m_deviceNum(deviceNum) 
               {
                  m_density           = 1.e3;
                  m_specificheat      = 3.8e3;
                  m_deltat            = 1.00;
                  m_bloodspecificheat = 3.4e3;
                  m_bodytemp          = 37.0;
                  m_x0          = 0.005;
                  m_y0          = 0.005;
                  m_z0          = 0.005;
               }
  // iterators for looping over element solution vector for each element
  typedef thrust::zip_iterator< thrust::tuple< 
     PetscScalarIter, PetscScalarIter, PetscScalarIter, PetscScalarIter
                       > > constitutive_iterator ; 
  constitutive_iterator ConstitutiveBegin() 
  {
    return thrust::make_zip_iterator(thrust::make_tuple( 
                     m_perfusion ->begin(),//0  perfusion
                     m_conduction->begin(),//1  conduction
                     m_scattering->begin(),//2  scattering
                     m_absorption->begin() //3  absorption
                                                       ));
  }
  constitutive_iterator ConstitutiveEnd() 
  {
    return thrust::make_zip_iterator(thrust::make_tuple( 
                     m_perfusion ->end(),//0  perfusion
                     m_conduction->end(),//1  conduction
                     m_scattering->end(),//2  scattering
                     m_absorption->end() //3  absorption
                                                       ));
  }
  // point_in_bbox from other post 
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple tuple) 
  { 
        // decode the hex node coordinates
        PetscScalar NodeXcoord[8], NodeYcoord[8], NodeZcoord[8] ;
        this->get_hex_vector(thrust::get<0>(thrust::get<0>(tuple)), NodeXcoord); 
        this->get_hex_vector(thrust::get<1>(thrust::get<0>(tuple)), NodeYcoord); 
        this->get_hex_vector(thrust::get<2>(thrust::get<0>(tuple)), NodeZcoord); 
  
        // decode local residual and solution
        PetscScalar ElementResidual[8], ElementSolution[8];
        this->get_hex_vector(thrust::get<1>(tuple), ElementResidual); 
        this->get_hex_vector(thrust::get<2>(tuple), ElementSolution); 
  
        // decode constitutive data
        PetscScalar Perfusion    = thrust::get<0>(thrust::get<3>(tuple));
        PetscScalar Conduction   = thrust::get<1>(thrust::get<3>(tuple));
        PetscScalar Absorption   = thrust::get<2>(thrust::get<3>(tuple));
        PetscScalar Scattering   = thrust::get<3>(thrust::get<3>(tuple));
  
        printf("rank=%d device=%d blockDim=(%d,%d,%d) gridDim=(%d,%d,%d) warpSize=%d blockIdx=(%d,%d,%d) threadIdx=(%d,%d,%d) node0=(%f,%f,%f) residual0=%f solution0=%f absorption=%f conduction=%f\n",m_rank,m_deviceNum,blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, warpSize,blockIdx.x,blockIdx.y,blockIdx.z,threadIdx.x,threadIdx.y,threadIdx.z,
                  NodeXcoord[0],NodeYcoord[0],NodeZcoord[0],
                  ElementResidual[0],ElementSolution[0],
                  Absorption     ,Conduction   );
        // //... do stuff with paramaters ... 
        //    thrust::get<0>(t) = sc * ( source
        //                   + m_density*m_specificheat/m_deltat* u_val 
        //                   + m_bloodspecificheat*m_perfusion*(m_bodytemp - 0.5*u_val) ) 
        // for (unsigned int qp=0; qp != n_qpoints; qp++)
        //   {
        //     // Compute the solution & its gradient at the old Newton iterate
        //     Number u_theta  = c.interior_value(   this->u_var,qp);
        //     Gradient grad_u = c.interior_gradient(this->u_var,qp);
  
        //     // get damage values
        //     Number  damage  = c.interior_value(   this->a_var,qp);
        //     Number DdamageDu= c.interior_value(   this->b_var,qp);
  
        //     Gradient DiffusionDirection = this->m_MathModel.DiffusionDirection(subdomain_id) ; 
        //     Gradient TempDiffusionDirection( 
        //              grad_u(0)*DiffusionDirection(0)  ,
        //              grad_u(1)*DiffusionDirection(1)  ,
        //              grad_u(2)*DiffusionDirection(2)  
        //                                    ); 
  
        //     // First, an i-loop over the velocity degrees of freedom.
        //     // We know that n_u_dofs == n_v_dofs so we can compute contributions
        //     // for both at the same time.
        //     for (unsigned int i=0; i != n_u_dofs; i++)
        //       {
        //         ElementResidual(i) += JxW[qp] * (
        //               phi[i][qp] * 
        //                (              // perfusion term (check the SIGN)
        //                 this->m_MathModel.PennesReactionTerm(field_id,u_theta,damage)
        //                          -    // source term 
        //                 this->m_MathModel.PennesSource(field_id,u_theta,
        //                                                damage,z_value, 
        //                                                qpoint[qp],
        //                                                this->m_PowerID)
        //                )
        //                          +    // diffusion term
        //               this->m_MathModel.ThermalConductivity(field_id,u_theta,damage) *
        //                                        ( TempDiffusionDirection * dphi[i][qp] )
        //       	) ;
        //         // convection term
        //         Fu(i) += JxW[qp] * phi[i][qp] * 
        //               ( this->m_MathModel.BulkFluidFlow(subdomain_id) * grad_u ) ; 
        //       }
        //   }
  } 

	// template <typename Tuple>
	// __host__ __device__
	// void operator()(Tuple t)
	// {
	// 	/* f = (2*u_i - u_(i+1) - u_(i-1))/h - h*exp(u_i) */
	//      thrust::get<0>(t) = 1;
        //      PetscInt Iz =  thrust::get<1>(t)/m_ym/m_xm;
        //      PetscInt Iy = (thrust::get<1>(t)-Iz*m_ym*m_xm)/m_xm;
        //      PetscInt Ix = (thrust::get<1>(t)-Iz*m_ym*m_xm- Iy*m_xm);
        //      PetscScalar sc      = m_hx*m_hz*m_hy;
        //      PetscScalar hxhzdhy = m_hx*m_hz/m_hy;
        //      PetscScalar hyhzdhx = m_hy*m_hz/m_hx;
        //      PetscScalar hxhydhz = m_hx*m_hy/m_hz;
        //      PetscScalar two     = 2.0;
        //      // print launch parameters and dbg info
        //      // printf("rank=%d device=%d blockDim=(%d,%d,%d) gridDim=(%d,%d,%d) warpSize=%d blockIdx=(%d,%d,%d) threadIdx=(%d,%d,%d) size=(%d,%d,%d) globalID=%d index=(%d,%d,%d)\n",m_rank,m_deviceNum,blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, warpSize,blockIdx.x,blockIdx.y,blockIdx.z,threadIdx.x,threadIdx.y,threadIdx.z,m_xm,m_ym,m_zm,thrust::get<8>(t),Ix,Iy,Iz);
        //      PetscScalar u_val       = thrust::get<0>(thrust::get<2>(t)) ;//1  u(i  ,j  ,k  )
        //      PetscScalar perfusion   = thrust::get<0>(thrust::get<3>(t)) ;//perfusion
        //      if (
        //          Ix > 0  && Ix < m_xm-1
        //                  &&
        //          Iy > 0  && Iy < m_ym-1
        //                  &&
        //          Iz > 0  && Iz < m_zm-1
        //         ) {
        //        // decode the tuple
        //        PetscScalar u_east      = thrust::get<1>(thrust::get<2>(t));//2  u(i+1,j  ,k  )
        //        PetscScalar u_west      = thrust::get<2>(thrust::get<2>(t));//3  u(i-1,j  ,k  )
        //        PetscScalar u_north     = thrust::get<3>(thrust::get<2>(t));//4  u(i  ,j+1,k  )
        //        PetscScalar u_south     = thrust::get<4>(thrust::get<2>(t));//5  u(i  ,j-1,k  )
        //        PetscScalar u_up        = thrust::get<5>(thrust::get<2>(t));//6  u(i  ,j  ,k+1)
        //        PetscScalar u_down      = thrust::get<6>(thrust::get<2>(t));//7  u(i  ,j  ,k-1)
        //        PetscScalar u_xx        = (-u_east  + two*u_val - u_west )*hyhzdhx;
        //        PetscScalar u_yy        = (-u_north + two*u_val - u_south)*hxhzdhy;
        //        PetscScalar u_zz        = (-u_up    + two*u_val - u_down )*hxhydhz;
        //        PetscScalar sqdist      = (m_hx * Ix - m_x0)*(m_hx * Ix - m_x0)
        //                                + (m_hy * Iy - m_y0)*(m_hy * Iy - m_y0)
        //                                + (m_hz * Iz - m_z0)*(m_hz * Iz - m_z0);
        //        PetscScalar source      = 1.e4 * exp(5.0/(sqdist +1.0));
        //        thrust::get<0>(t) = sc * ( source
        //                       + m_density*m_specificheat/m_deltat* u_val 
        //                       + m_bloodspecificheat*m_perfusion*(m_bodytemp - 0.5*u_val) ) 
        //                       + m_conduction/2.0* (u_xx + u_yy + u_zz) ;
        //      } else { // dirichlet bc everywhere else
        //        thrust::get<0>(t) = u_val;
        //      } 
	// 	
	// }
};
int main(int argc,char **argv) 
{
  SNES           snes; 
  Vec            x,f;  
  Mat            J;
  PetscErrorCode ierr;
  hipError_t      ierrCuda;
  char           *tmp,typeName[256];
  int            myrank;
  PetscBool      flg;

  PetscInitialize(&argc,&argv,(char *)0,help);

  MPI_Comm_rank(PETSC_COMM_WORLD, &myrank);
  int deviceNum=myrank;
  {
    int deviceCount;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
    
    ierr = PetscPrintf(PETSC_COMM_SELF, "!!!!!found %d devices !!!!!\n",deviceCount);CHKERRQ(ierr);
    if (deviceCount == 0) {
      ierr = PetscPrintf(PETSC_COMM_SELF, "!!!!!No devices found!!!!!\n");CHKERRQ(ierr);
      return -1000;
    }

    if (deviceNum >= deviceCount || deviceNum < 0) {
      ierr = PetscPrintf(PETSC_COMM_SELF, "\n!!!!!Invalid GPU number %d given hence default gpu %d will be used !!!!!\n", deviceNum, 0);CHKERRQ(ierr);
      deviceNum = 0;
    }
  }

  ierrCuda =  hipSetDevice(deviceNum);
  if (ierrCuda != hipSuccess) {
    ierr = PetscPrintf(PETSC_COMM_SELF, " cuda Error: %s , exiting\n",hipGetErrorString( ierrCuda));CHKERRQ(ierr);
    return -1;
  }
  ierr = PetscPrintf(PETSC_COMM_SELF, " reseting GPU: \n");CHKERRQ(ierr);
  CUDA_SAFE_CALL(hipDeviceReset());

  ierr = PetscPrintf(PETSC_COMM_SELF, "Running on...\n\n");CHKERRQ(ierr);
  hipDeviceProp_t deviceProp;
  if (hipGetDeviceProperties(&deviceProp, deviceNum) == hipSuccess) {
    ierr = PetscPrintf(PETSC_COMM_SELF, " Device %d: %s %d.%d\n", deviceNum, deviceProp.name,deviceProp.major,deviceProp.minor);CHKERRQ(ierr);
    ierr = PetscPrintf(PETSC_COMM_SELF," Global memory available on device in bytes %d\n"                            ,  deviceProp.totalGlobalMem                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Shared memory available per block in bytes %d\n"                            ,  deviceProp.sharedMemPerBlock               );
    ierr = PetscPrintf(PETSC_COMM_SELF," 32-bit registers available per block %d\n"                                  ,  deviceProp.regsPerBlock                    );
    ierr = PetscPrintf(PETSC_COMM_SELF," Warp size in threads %d\n"                                                  ,  deviceProp.warpSize                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum pitch in bytes allowed by memory copies %d\n"                       ,  deviceProp.memPitch                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum number of threads per block %d\n"                                   ,  deviceProp.maxThreadsPerBlock              );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[0]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[1]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a block %d\n"                             ,  deviceProp.maxThreadsDim[2]                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[0]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[1]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum size of each dimension of a grid %d\n"                              ,  deviceProp.maxGridSize[2]                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Clock frequency in kilohertz %d\n"                                          ,  deviceProp.clockRate                       );
    ierr = PetscPrintf(PETSC_COMM_SELF," Constant memory available on device in bytes %d\n"                          ,  deviceProp.totalConstMem                   );
    ierr = PetscPrintf(PETSC_COMM_SELF," Alignment requirement for textures %d\n"                                    ,  deviceProp.textureAlignment                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Number of multiprocessors on device %d\n"                                   ,  deviceProp.multiProcessorCount             );
    ierr = PetscPrintf(PETSC_COMM_SELF," Specified whether there is a run time limit on kernels %d\n"                ,  deviceProp.kernelExecTimeoutEnabled        );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device is integrated as opposed to discrete %d\n"                           ,  deviceProp.integrated                      );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device can map host memory with hipHostAlloc/hipHostGetDevicePointer %d\n",  deviceProp.canMapHostMemory                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Compute mode (See ::hipComputeMode) %d\n"                                  ,  deviceProp.computeMode                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D texture size %d\n"                                               ,  deviceProp.maxTexture1D                    );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D texture dimensions %d\n"                                         ,  deviceProp.maxTexture2D[0]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D texture dimensions %d\n"                                         ,  deviceProp.maxTexture2D[1]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[0]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[1]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 3D texture dimensions %d\n"                                         ,  deviceProp.maxTexture3D[2]                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture1DLayered[0]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 1D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture1DLayered[1]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[0]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[1]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum 2D layered texture dimensions %d\n"                                 ,  deviceProp.maxTexture2DLayered[2]          );
    ierr = PetscPrintf(PETSC_COMM_SELF," Alignment requirements for surfaces %d\n"                                   ,  deviceProp.surfaceAlignment                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device can possibly execute multiple kernels concurrently %d\n"             ,  deviceProp.concurrentKernels               );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device has ECC support enabled %d\n"                                        ,  deviceProp.ECCEnabled                      );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI bus ID of the device %d\n"                                              ,  deviceProp.pciBusID                        );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI device ID of the device %d\n"                                           ,  deviceProp.pciDeviceID                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," PCI domain ID of the device %d\n"                                           ,  deviceProp.pciDomainID                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," 1 if device is a Tesla device using TCC driver, 0 otherwise %d\n"           ,  deviceProp.tccDriver                       );
    ierr = PetscPrintf(PETSC_COMM_SELF," Number of asynchronous engines %d\n"                                        ,  deviceProp.asyncEngineCount                );
    ierr = PetscPrintf(PETSC_COMM_SELF," Device shares a unified address space with the host %d\n"                   ,  deviceProp.unifiedAddressing               );
    ierr = PetscPrintf(PETSC_COMM_SELF," Peak memory clock frequency in kilohertz %d\n"                              ,  deviceProp.memoryClockRate                 );
    ierr = PetscPrintf(PETSC_COMM_SELF," Global memory bus width in bits %d\n"                                       ,  deviceProp.memoryBusWidth                  );
    ierr = PetscPrintf(PETSC_COMM_SELF," Size of L2 cache in bytes %d\n"                                             ,  deviceProp.l2CacheSize                     );
    ierr = PetscPrintf(PETSC_COMM_SELF," Maximum resident threads per multiprocessor %d\n"                           ,  deviceProp.maxThreadsPerMultiProcessor     );
  } else {
    ierr = PetscPrintf(PETSC_COMM_SELF, " Unable to determine device %d properties, exiting\n",deviceNum);CHKERRQ(ierr);
    ierr = PetscPrintf(PETSC_COMM_SELF, " cuda Error: %s , exiting\n",hipGetErrorString( ierrCuda));CHKERRQ(ierr);
    return -1;
  }

  PetscLogEventRegister("ComputeFunction",0,&LogFunction); 
  ierr = PetscOptionsGetString(PETSC_NULL,"-da_vec_type",typeName,256,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = PetscStrstr(typeName,"cusp",&tmp);CHKERRQ(ierr);
    if (tmp) useCUSP = PETSC_TRUE;
  }

  size_t sizeIndex = 3 * sizeof(PetscInt);
  CUDA_SAFE_CALL(hipMalloc((void **) &cudaTest, sizeIndex));   // Allocate array on device

  //ierr = DMDACreate1d(PETSC_COMM_WORLD,DMDA_BOUNDARY_NONE,-8,1,1,PETSC_NULL,&da);CHKERRQ(ierr);
  PetscInt globalSize = 125;
  globalSize = 99;
  DM             da;
  ierr = DMDACreate3d(PETSC_COMM_WORLD,DMDA_BOUNDARY_NONE,DMDA_BOUNDARY_NONE,DMDA_BOUNDARY_NONE,DMDA_STENCIL_STAR,-globalSize,-globalSize,-globalSize,PETSC_DECIDE,PETSC_DECIDE,PETSC_DECIDE,1,1,PETSC_NULL,PETSC_NULL,PETSC_NULL,&da);CHKERRQ(ierr);
  ierr = DMCreateGlobalVector(da,&x); VecDuplicate(x,&f);CHKERRQ(ierr);
  if (useCUSP)
    {
     ierr = DMCreateMatrix(da,MATAIJCUSP,&J);CHKERRQ(ierr);
    }
  else
    {
     ierr = DMCreateMatrix(da,MATAIJ,&J);CHKERRQ(ierr);
    }

  PetscInt       GlobalDAMx,GlobalDAMy,GlobalDAMz;
  ierr = DMDAGetInfo(da,PETSC_IGNORE,&GlobalDAMx,&GlobalDAMy,&GlobalDAMz,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
  // ierr = DMDAGetCorners(da,&xs,&ys,&zs,&xm,&ym,&zm);CHKERRQ(ierr);
  WFSModel FemModel(myrank,deviceNum,100);// fem mesh data
  // StarStencil  stencil_op(0,0,xs,ys,zs,xm,ym,zm,hx,hy,hz);// transformation operator
  // ierr = DMSetApplicationContext(da,&stencil_op);CHKERRQ(ierr);

  ierr = SNESCreate(PETSC_COMM_WORLD,&snes);CHKERRQ(ierr);
  ierr = SNESSetFunction(snes,f,ComputeFunction,da);CHKERRQ(ierr);
  ierr = SNESSetFromOptions(snes);CHKERRQ(ierr);
  for (PetscInt iii = 0 ; iii < 1 ; iii++)
    {
     ierr = PetscPrintf(PETSC_COMM_WORLD, "gpu check %d \n",iii);CHKERRQ(ierr);
     ierr = ComputeFunction(snes,x,f,(void *)da);
    }
  ierr = SNESSolve(snes,PETSC_NULL,x);CHKERRQ(ierr);

  ierr = MatDestroy(&J);CHKERRQ(ierr);
  ierr = VecDestroy(&x);CHKERRQ(ierr);
  ierr = VecDestroy(&f);CHKERRQ(ierr);
  ierr = SNESDestroy(&snes);CHKERRQ(ierr);
  ierr = DMDestroy(&da);CHKERRQ(ierr);

  // call device reset to flush buffer
  CUDA_SAFE_CALL(hipDeviceReset());
  PetscFinalize();
  return 0;
}


// PetscErrorCode ComputeFunction(SNES snes,Vec u,Vec f,void *ctx) 
// {
//   PetscInt       i,j,k;
//   PetscInt       ustartshift,uendshift,xoffset,yoffset,zoffset,fstart;
//   PetscScalar    ***uu,***ff,hxhzdhy,hyhzdhx,hxhydhz;
//   PetscScalar    u_val,u_east,u_west,u_north,u_south,u_up, u_down, u_xx, u_yy,u_zz,sc ,two =2.0;
//   DM             da = (DM) ctx; 
//   Vec            ulocal;
//   PetscErrorCode ierr;
//   PetscMPIInt    rank,size;
//   MPI_Comm       comm;
//   CUSPARRAY      *uarray,*farray;
//   PetscLogEventBegin(LogFunction,0,0,0,0); // init libMesh
// 
//   ierr = DMGetLocalVector(da,&ulocal);CHKERRQ(ierr);
//   ierr = DMGlobalToLocalBegin(da,u,INSERT_VALUES,ulocal);CHKERRQ(ierr);
//   ierr = DMGlobalToLocalEnd(da,u,INSERT_VALUES,ulocal);CHKERRQ(ierr);
//   StarStencil  *stencil_op;
//   ierr = DMGetApplicationContext(da,(void *)&stencil_op);CHKERRQ(ierr);
//   hxhzdhy = stencil_op->m_hx*stencil_op->m_hz/stencil_op->m_hy;
//   hyhzdhx = stencil_op->m_hy*stencil_op->m_hz/stencil_op->m_hx;
//   hxhydhz = stencil_op->m_hx*stencil_op->m_hy/stencil_op->m_hz;
//   sc      = stencil_op->m_hx*stencil_op->m_hy*stencil_op->m_hz*3.0;
// 
//   if (useCUSP) {
//     ierr = VecCUSPGetArrayRead(ulocal,&uarray);CHKERRQ(ierr);
//     ierr = VecCUSPGetArrayWrite(f,&farray);CHKERRQ(ierr);
//     ierr = PetscObjectGetComm((PetscObject)da,&comm);CHKERRQ(ierr);
//     ierr = MPI_Comm_size(comm,&size);CHKERRQ(ierr);
//     ierr = MPI_Comm_rank(comm,&rank);CHKERRQ(ierr);
//     if (rank) ustartshift = 1; else ustartshift = 0;
//     if (rank != size-1) uendshift = 1; else uendshift = 0;
//     xoffset = 1;
//     yoffset = stencil_op->m_xm;
//     zoffset = stencil_op->m_xm*stencil_op->m_ym;
//     ierr = VecGetOwnershipRange(f,&fstart,PETSC_NULL);CHKERRQ(ierr);
//     try {
//       
//       // typedef these iterators for shorthand
//       thrust::for_each(
// 		       thrust::make_zip_iterator(
// 						 thrust::make_tuple(
//             farray->begin(),                              //0
//             thrust::counting_iterator<int>(fstart) ,       //1
// 		       thrust::make_zip_iterator(
// 						 thrust::make_tuple(
//             uarray->begin()+ustartshift,                  //1  u(i  ,j  ,k  )
//             uarray->begin()+ustartshift + xoffset,        //2  u(i+1,j  ,k  )
//             uarray->begin()+ustartshift - xoffset,        //3  u(i-1,j  ,k  )
//             uarray->begin()+ustartshift + yoffset,        //4  u(i  ,j+1,k  )
//             uarray->begin()+ustartshift - yoffset,        //5  u(i  ,j-1,k  )
//             uarray->begin()+ustartshift + zoffset,        //6  u(i  ,j  ,k+1)
//             uarray->begin()+ustartshift - zoffset         //7  u(i  ,j  ,k-1)
//                                                                     )), 
// 		       thrust::make_zip_iterator(
// 						 thrust::make_tuple(
//             thrust::constant_iterator<PetscScalar>(6.0  ),//0  perfusion
//             thrust::constant_iterator<PetscScalar>(0.57 ),//1  conduction
//             thrust::constant_iterator<PetscScalar>(5.e2 ),//2  scattering
//             thrust::constant_iterator<PetscScalar>(14.e3) //3  absorption
//                                                                     )) 
//                                                                     )), 
// 		       thrust::make_zip_iterator(
// 						 thrust::make_tuple(
//             farray->end(),                                            //0
//             thrust::counting_iterator<int>(fstart) + u->map->n ,      //1
// 		       thrust::make_zip_iterator(
// 						 thrust::make_tuple(
//             uarray->end()+uendshift,                  //2_0  u(i  ,j  ,k  )
//             uarray->end()+uendshift + xoffset,        //2_1  u(i+1,j  ,k  )
//             uarray->end()+uendshift - xoffset,        //2_2  u(i-1,j  ,k  )
//             uarray->end()+uendshift + yoffset,        //2_3  u(i  ,j+1,k  )
//             uarray->end()+uendshift - yoffset,        //2_4  u(i  ,j-1,k  )
//             uarray->end()+uendshift + zoffset,        //2_5  u(i  ,j  ,k+1)
//             uarray->end()+uendshift - zoffset         //2_6  u(i  ,j  ,k-1)
//                                                                     )), 
// 		       thrust::make_zip_iterator(
// 						 thrust::make_tuple(
//             thrust::constant_iterator<PetscScalar>(6.0  ),//3_0  perfusion
//             thrust::constant_iterator<PetscScalar>(0.57 ),//3_1  conduction
//             thrust::constant_iterator<PetscScalar>(5.e2 ),//3_2  scattering
//             thrust::constant_iterator<PetscScalar>(14.e3) //3_3  absorption
//                                                                     ))  
//                                                                     )),
// 		       *stencil_op);
//       
//       PetscInt hostTest[3]={-1,-1,-1};
//       //CUDA_SAFE_CALL(hipMemcpy(hostTest, cudaTest,3*sizeof(PetscInt),hipMemcpyDeviceToHost));
//       ierr = PetscPrintf(PETSC_COMM_WORLD, "%d %d %d \n",hostTest[0],hostTest[1],hostTest[2]);CHKERRQ(ierr);
//     }
//     catch(char* all){
//       ierr = PetscPrintf(PETSC_COMM_WORLD, "Thrust is not working\n");CHKERRQ(ierr);
//     }
//     ierr = VecCUSPRestoreArrayRead(ulocal,&uarray);CHKERRQ(ierr);
//     ierr = VecCUSPRestoreArrayWrite(f,&farray);CHKERRQ(ierr);
//   } else {
//     ierr = DMDAVecGetArray(da,ulocal,&uu);CHKERRQ(ierr);
//     ierr = DMDAVecGetArray(da,f,&ff);CHKERRQ(ierr);
//     
//     PetscInt       GlobalDAMx,GlobalDAMy,GlobalDAMz;
//     ierr = DMDAGetInfo(da,PETSC_IGNORE,&GlobalDAMx,&GlobalDAMy,&GlobalDAMz,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
//     /* Compute function over the locally owned part of the grid */
//     for (k=stencil_op->m_zs; k<stencil_op->m_zs+stencil_op->m_zm; k++) {
//       for (j=stencil_op->m_ys; j<stencil_op->m_ys+stencil_op->m_ym; j++) {
//         for (i=stencil_op->m_xs; i<stencil_op->m_xs+stencil_op->m_xm; i++) {
//           if (i == 0 || j == 0 || k == 0 || i == GlobalDAMx-1 || j == GlobalDAMy-1 || k == GlobalDAMz-1) {
//             ff[k][j][i] = uu[k][j][i];
//           } else {
//             u_val       = uu[k][j][i];
//             u_east      = uu[k][j][i+1];
//             u_west      = uu[k][j][i-1];
//             u_north     = uu[k][j+1][i];
//             u_south     = uu[k][j-1][i];
//             u_up        = uu[k+1][j][i];
//             u_down      = uu[k-1][j][i];
//             u_xx        = (-u_east  + two*u_val - u_west )*hyhzdhx;
//             u_yy        = (-u_north + two*u_val - u_south)*hxhzdhy;
//             u_zz        = (-u_up    + two*u_val - u_down )*hxhydhz;
//             ff[k][j][i]  = u_xx + u_yy + u_zz - sc*PetscExpScalar(u_val);
//           }
//         }
//       }
//     }
//     ierr = DMDAVecRestoreArray(da,ulocal,&uu);CHKERRQ(ierr);
//     ierr = DMDAVecRestoreArray(da,f,&ff);CHKERRQ(ierr);
//   }
//   ierr = DMRestoreLocalVector(da,&ulocal);CHKERRQ(ierr);
//   PetscLogEventEnd(LogFunction,0,0,0,0);   // init libMesh
//   //VecView(u,0);printf("f\n");
//   //VecView(f,0);
//   return 0;
// 
// }
// PetscErrorCode ComputeJacobian(SNES snes,Vec x,Mat *J,Mat *B,MatStructure *flag,void *ctx)
// {
//   DM             da = (DM) ctx; 
//   Vec            xlocal;
//   PetscErrorCode ierr;
//   if(jacobianComputed) return 0;
//   jacobianComputed = PETSC_TRUE;
// 
//   ierr = DMGetLocalVector(da,&xlocal);DMGlobalToLocalBegin(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
//   ierr = DMGlobalToLocalEnd(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
// 
//   PetscInt       GlobalDAMx,GlobalDAMy,GlobalDAMz,xs,xm,ys,ym,zs,zm;
//   PetscScalar    hx,hy,hz;
//   ierr = DMDAGetInfo(da,PETSC_IGNORE,&GlobalDAMx,&GlobalDAMy,&GlobalDAMz,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
//   hx     = 1.0/(PetscReal)(GlobalDAMx-1);
//   hy     = 1.0/(PetscReal)(GlobalDAMy-1);
//   hz     = 1.0/(PetscReal)(GlobalDAMz-1);
//   ierr = DMDAGetCorners(da,&xs,&ys,&zs,&xm,&ym,&zm);CHKERRQ(ierr);
// 
//   PetscScalar    hxhzdhy,hyhzdhx,hxhydhz,sc;
//   hxhzdhy = hx*hz/hy;
//   hyhzdhx = hy*hz/hx;
//   hxhydhz = hx*hy/hz;
//   sc      = hx*hy*hz*3.0;
// 
//   ierr = MatZeroEntries(*J);CHKERRQ(ierr);
//   ierr = MatShift(*J,1.0);CHKERRQ(ierr);
// 
//   StarStencil  *stencil_op;
//   ierr = DMGetApplicationContext(da,(void *)&stencil_op);CHKERRQ(ierr);
// 
//   /* Compute function over the locally owned part of the grid */
//   PetscScalar    v[7],two = 2.0;
//   MatStencil     col[7],row;
//   PetscInt       i,j,k;
//   for (k=zs; k<zs+zm; k++) {
//     for (j=ys; j<ys+ym; j++) {
//       for (i=xs; i<xs+xm; i++) {
//         row.k = k; row.j = j; row.i = i;
//         if (i > 0 && j > 0 && k > 0 && i < GlobalDAMx-1 && j < GlobalDAMy-1 && k < GlobalDAMz-1) {
//           v[0] = -0.5 * stencil_op->m_conduction * hxhydhz; col[0].k=k-1;col[0].j=j;  col[0].i = i;
//           v[1] = -0.5 * stencil_op->m_conduction * hxhzdhy; col[1].k=k;  col[1].j=j-1;col[1].i = i;
//           v[2] = -0.5 * stencil_op->m_conduction * hyhzdhx; col[2].k=k;  col[2].j=j;  col[2].i = i-1;
//           v[3] =  sc*(  stencil_op->m_density*stencil_op->m_specificheat/stencil_op->m_deltat 
//                     + 0.5 * stencil_op->m_perfusion * stencil_op->m_bloodspecificheat) 
//                     + 1.0 * stencil_op->m_bloodspecificheat * (hyhzdhx+hxhzdhy+hxhydhz);
//                            col[3].k=row.k;col[3].j=row.j;col[3].i = row.i;
//           v[4] = -0.5 * stencil_op->m_conduction * hyhzdhx; col[4].k=k;  col[4].j=j;  col[4].i = i+1;
//           v[5] = -0.5 * stencil_op->m_conduction * hxhzdhy; col[5].k=k;  col[5].j=j+1;col[5].i = i;
//           v[6] = -0.5 * stencil_op->m_conduction * hxhydhz; col[6].k=k+1;col[6].j=j;  col[6].i = i;
//           ierr = MatSetValuesStencil(*J,1,&row,7,col,v,INSERT_VALUES);CHKERRQ(ierr);
//         }
//       }
//     }
//   }
// 
//   ierr = MatAssemblyBegin(*J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
//   ierr = MatAssemblyEnd(*J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
//   *flag = SAME_NONZERO_PATTERN;
//   ierr = DMRestoreLocalVector(da,&xlocal);CHKERRQ(ierr);
//   return 0;
// }

// gNek is really lightweight - and the input requirements are clearly defined.
// Feed us mesh vertex coordinates, the element-vertex connectivity, the 
// element boundary conditions and the material parameters and we can
// feed back a solution. This can even be done through cubit or gmsh files [ I think ].
// 
// I have to say that GPU compute is pretty much all or nothing - we also try to 
// avoid too much traffic between host and device. However, we do the 
// preprocessing on the host as this is usually a sub-dominant cost.
PetscErrorCode ComputeFunction(SNES snes,Vec u,Vec f,void *ctx) 
{
  WFSModel *FemModel= (WFSModel*) ctx;
  PetscErrorCode ierr;
  // get solution array for reading
  // FIXME: will not work for mpi distributed array
  // TODO:  fix for MPI
  ierr = VecCUSPGetArrayRead(u,&FemModel->uarray);CHKERRQ(ierr);
  // get residual array for writing
  ierr = VecCUSPGetArrayWrite(f,&FemModel->farray);CHKERRQ(ierr);

  // loop over elements
  // ensure thread safety by each thread writing to its own local residual
  // ie similar to DG methods
  thrust::for_each( 
     thrust::make_zip_iterator(thrust::make_tuple(
                               FemModel->ElementBegin(),
                               FemModel->ResidualBegin(),
                               FemModel->SolutionBegin(), 
                               FemModel->ConstitutiveBegin()
                              )), 
     thrust::make_zip_iterator(thrust::make_tuple(
                               FemModel->ElementEnd(),
                               FemModel->ResidualEnd(),
                               FemModel->SolutionEnd(), 
                               FemModel->ConstitutiveEnd()
                              )), 
     *FemModel // call the overloaded operator() from this class
                  ); 
  // Reduce the expanded residual to the usual 
  // continuous additive contributions
  // first need to sort
  typedef CUSPARRAY::iterator PetscScalarIter;
  typedef CUSPINTARRAYGPU::iterator    PetscIntIter;
  //thrust::sort_by_key<PetscIntIter,PetscScalarIter>(
  //                           FemModel->m_LocalElementMap->begin(),
  //                           FemModel->m_LocalElementMap->end(),
  //                           FemModel->element_residuals->begin()
  //                          );
  // reduce the sorted array
  thrust::reduce_by_key(
                        FemModel->m_LocalElementMap->begin(),
                        FemModel->m_LocalElementMap->end(),
                        FemModel->element_residuals->begin(),
                        thrust::make_discard_iterator(),
                        FemModel->farray->begin()
                       );
     
  return 0;
}

